#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <iostream>
#include <fstream>
#include <iterator>
#include <experimental/filesystem>
#include "binFile.h"
#include "linkList.h"

namespace fs = std::experimental::filesystem;

std::string getName(std::string name) {
	int ind = 0;
	for (int i = 0; i < name.length(); i++) {
		if (name.at(i) == '\\')
			ind = i;
	}
	return name.substr(ind + 1, name.length());
}

void getFile(bool isEncryptedFile, binFile & file, std::string path) {
	std::ifstream infile(path, std::ios::binary);
	infile.seekg(0, infile.end);
	int length = infile.tellg();
	infile.seekg(0, infile.beg);

	file.setName(getName(path));
	char* temp = new char[length];
	infile.read(temp, length);
	if (isEncryptedFile) {
		unsigned char t = temp[0];
		length = 0x00000000 | (unsigned int)t << 24;
		t = temp[1];
		length = length | (unsigned int)t << 16;
		t = temp[2];
		length = length | (unsigned int)t << 8;
		t = temp[3];
		length = length | (unsigned int)t;

		temp = &temp[4];
	}
	file.setFile(temp, length);
}

int getFileNum(std::string path) {
	int re = 0;
	for (auto& entry : fs::directory_iterator(path))
		re++;
	return re;
}

void getFiles(bool isEncryptedFile, binFile * &files, int fileNum, std::string path) {
	files = new binFile[fileNum];

	size_t ind = 0;
	for (const auto& entry : fs::directory_iterator(path)) {
		std::ifstream infile(entry.path(), std::ios::binary);
		infile.seekg(0, infile.end);
		int length = infile.tellg();
		infile.seekg(0, infile.beg);

		files[ind].setName(getName(entry.path().u8string()));
		char* temp = new char[length];
		infile.read(temp, length);
		if (isEncryptedFile) {
			unsigned char t = temp[0];
			length = 0x00000000 | (unsigned int)t << 24;
			t = temp[1];
			length = length | (unsigned int)t << 16;
			t = temp[2];
			length = length | (unsigned int)t << 8;
			t = temp[3];
			length = length | (unsigned int)t;

			temp = &temp[4];
		}
		files[ind].setFile(temp, length);
		ind++;
	}
}

void writeFiles(bool isEncrypt, binFile * files, int fileNum, std::string path) {
	for (int i = 0; i < fileNum; i++) {
		std::ofstream file;
		char* temp = files[i].getFile();
		file.open((path + files[i].getName()), std::ios::out | std::ios::binary);

		if (isEncrypt) {
			char* t = new char[4];
			int s = files[i].getSize();
			t[0] = s >> 24;
			t[1] = (s << 8) >> 24;
			t[2] = (s << 16) >> 24;
			t[3] = (s << 24) >> 24;

			for (int p = 0; p < 4; p++)
				file << t[p];
		}
		for (int p = 0; p < files[i].getSize(); p++)
			file << temp[p];

		file.close();
	}

}

void writeFile(bool isEncrypt, binFile files, std::string path) {
	std::ofstream file;
	file.open((path + files.getName()), std::ios::out | std::ios::binary);
	char* temp = files.getFile();
	if (isEncrypt) {
		char* t = new char[4];
		int s = files.getSize();
		t[0] = s >> 24;
		t[1] = (s << 8) >> 24;
		t[2] = (s << 16) >> 24;
		t[3] = (s << 24) >> 24;

		for (int p = 0; p < 4; p++)
			file << t[p];

		for (int p = 0; p < files.getSize(); p++)
			file << temp[p];

	}
	else {
		for (int p = 0; p < files.getSize(); p++)
			file << temp[p];

	}

	file.close();
}

__int32 genSequence(bool flipFirst, bool flipLast, __int16 firstID, __int16 lastID)
{
	int re = 0x00000000;
	re = re | (unsigned int)flipFirst << 31 | (unsigned int)firstID << 16 | (unsigned int)flipLast << 15 | (unsigned int)lastID;
	return re;
}

__int32** rnGen(std::string seed, int blockNum, int enLevel, int& sLength) {
	srand(1);
	int s = 0;
	for (int i = 0; i < seed.length(); i++)
		s += (int)seed.at(i) * rand();

	__int32** sequence = new __int32*[enLevel];
	if (blockNum % 2 == 1)
		sLength = (blockNum - 1) / 2;
	else
		sLength = blockNum / 2;

	srand(s * rand());
	for (int p = 0; p < enLevel; p++) {
		sequence[p] = new __int32[sLength];
		linkList* list = new linkList;
		for (int i = 0; i < blockNum; i++)
			list->add(i);

		for (int i = 0; i < sLength; i++) {
			sequence[p][i] = genSequence(rand() % 2 == 1, rand() % 2 == 1,
				(__int16)list->getContent(((float)rand() / RAND_MAX) * list->getSize() + 1),
				(__int16)list->getContent(((float)rand() / RAND_MAX) * list->getSize() + 1));
		}
	}
	return sequence;
}

__global__ void d_encrypt(char* array, __int32* sequence) {
	int blockID = blockIdx.x;
	__int32 s = sequence[blockID];

	__int16 ID1 = threadIdx.x + ((s << 1) >> 17)*blockDim.x;
	__int16 ID2 = threadIdx.x + ((s << 17) >> 17)*blockDim.x;
	bool flip1 = (s >> 31);
	bool flip2 = (s << 16) >> 31;

	if (flip1 == flip2)
	{
		array[ID1] = array[ID1] ^ array[ID2];
	}
	else if (flip1 && !flip2)
	{
		array[ID1] = (~array[ID1]) ^ array[ID2];
	}
	else if (!flip1 && flip2)
	{
		array[ID1] = array[ID1] ^ (~array[ID2]);
	}
}

__global__ void d_decrypt(char* array, __int32* sequence) {
	int blockID = blockIdx.x;
	__int32 s = sequence[blockID];

	__int16 ID1 = threadIdx.x + ((s << 1) >> 17) * blockDim.x;
	__int16 ID2 = threadIdx.x + ((s << 17) >> 17) * blockDim.x;
	bool flip1 = (s >> 31);
	bool flip2 = (s << 16) >> 31;

	if (flip1 == flip2)
	{
		array[ID1] = array[ID1] ^ array[ID2];
	}
	else if (flip1 && !flip2)
	{
		array[ID1] = array[ID1] ^ (~array[ID2]);
	}
	else if (!flip1 && flip2)
	{
		array[ID1] = (~array[ID1]) ^ array[ID2];
	}
}

void encrypt(binFile& file, int blockLength, __int32** sequence, int enLevel, int slength) {
	for (int i = 0; i < enLevel; i++) {
		__int32* d_s = NULL;
		char* d_bytes = NULL;

		dim3 block(blockLength);
		dim3 grid(slength);

		hipMalloc((__int32**)& d_s, sizeof(__int32) * slength);
		hipMalloc((char**)& d_bytes, sizeof(char) * file.getSize());
		hipMemcpy(d_s, sequence[i], slength * sizeof(__int32), hipMemcpyHostToDevice);
		hipMemcpy(d_bytes, file.getFile(), file.getSize() * sizeof(char), hipMemcpyHostToDevice);

		d_encrypt <<< grid, block >>> (d_bytes, d_s);

		char* h_temp;
		h_temp = (char*)malloc(sizeof(char) * file.getSize());
		hipMemcpy(h_temp, d_bytes, file.getSize() * sizeof(char), hipMemcpyDeviceToHost);
		file.setFile(h_temp, file.getSize());
		
		hipFree(d_s);
		hipFree(d_s);
		free(h_temp);
	}
}

void decrypt(binFile& file, int blockLength, __int32** sequence, int enLevel, int slength) {
	for (int i = enLevel - 1; i >= 0; i--) {
		__int32* d_s = NULL;
		char* d_bytes = NULL;

		dim3 block(blockLength);
		dim3 grid(slength);

		hipMalloc((__int32 **)& d_s, sizeof(__int32) * slength);
		hipMalloc((char**)& d_bytes, sizeof(char) * file.getSize());
		hipMemcpy(d_s, sequence[i], slength * sizeof(__int32), hipMemcpyHostToDevice);
		hipMemcpy(d_bytes, file.getFile(), file.getSize() * sizeof(char), hipMemcpyHostToDevice);

		d_decrypt <<< grid, block >>> (d_bytes, d_s);

		char* h_temp;
		h_temp = (char*)malloc(sizeof(char) * file.getSize());
		hipMemcpy(h_temp, d_bytes, file.getSize() * sizeof(char), hipMemcpyDeviceToHost);
		file.setFile(h_temp, file.getSize());

		hipFree(d_s);
		hipFree(d_s);
		free(h_temp);
	}
}

void deSequence(bool& flipFirst, bool& flipLast, __int16& firstID, __int16& lastID, __int32 sequence) {
	firstID = ((sequence << 1) >> 17);
	lastID = ((sequence << 17) >> 17);
	flipFirst = (sequence >> 31);
	flipLast = (sequence << 16) >> 31;
}
void encrypt_leg(char** array, int chunkDim, __int32 sequence) {
	__int16 chunk1ID, chunk2ID;
	bool flip1, flip2;
	deSequence(flip1, flip2, chunk1ID, chunk2ID, sequence);
	char* arr1 = array[chunk1ID];
	char* arr2 = array[chunk2ID];

	if (flip1 == flip2)
	{
		for (int i = 0; i < chunkDim; i++)
			arr1[i] = arr1[i] ^ arr2[i];
	}
	else if (flip1 && !flip2)
	{
		for (int i = 0; i < chunkDim; i++)
			arr1[i] = (~arr1[i]) ^ arr2[i];
	}
	else if (!flip1 && flip2)
	{
		for (int i = 0; i < chunkDim; i++)
			arr1[i] = arr1[i] ^ (~arr2[i]);
	}
}
void decrypt_leg(char** array, int chunkDim, __int32 sequence) {
	__int16 chunk1ID, chunk2ID;
	bool flip1, flip2;
	deSequence(flip1, flip2, chunk1ID, chunk2ID, sequence);
	char* arr1 = array[chunk1ID];
	char* arr2 = array[chunk2ID];

	if (flip1 == flip2)
	{
		for (int i = chunkDim - 1; i >= 0; i--)
			arr1[i] = arr1[i] ^ arr2[i];
	}
	else if (flip1 && !flip2)
	{
		for (int i = chunkDim - 1; i >= 0; i--)
			arr1[i] = arr1[i] ^ (~arr2[i]);
	}
	else if (!flip1 && flip2)
	{
		for (int i = chunkDim - 1; i >= 0; i--)
			arr1[i] = (~arr1[i]) ^ arr2[i];
	}
}

void test()
{

	std::string pass = "1";
	binFile file;
	getFile(false, file, "C:\\Users\\george\\Desktop\\1.txt");
	int slength;
	int blockDim = 2;
	int enlevel = 4;
	int blockNum = (file.getSize() + blockDim - 1) / blockDim;
	std::cout << blockNum << std::endl;
	__int32** s = rnGen(pass, blockNum, enlevel, slength);
	std::cout << slength << std::endl;

	char** c = new char* [blockNum];
	int ind = 0;
	char* f = file.getFile();
	for (int i = 0; i < blockNum; i++) {
		c[i] = new char[blockDim];
		for (int p = 0; p < blockDim; p++) {
			c[i][p] = f[ind];
			ind++;
		}
	}
	for (int i = 0; i < slength; i++)
		encrypt_leg(c, blockDim, s[0][i]);
	for (int i = slength - 1; i >= 0; i--)
		decrypt_leg(c, blockDim, s[0][i]);

	encrypt(file, blockDim, s, enlevel, slength);
	decrypt(file, blockDim, s, enlevel, slength);

	char* experimant = file.getFile();
	char* con = new char[file.getSize()];
	for (int i = 0; i < file.getSize(); i++)
		con[i] = c[i / blockDim][i % blockDim];

	for (int i = 0; i < slength; i++)
		std::cout << std::hex << s[0][i] << std::endl;

	for (int i = 0; i < file.getSize(); i++) {
		//	if (experimant[i] != con[i])
		std::cout << "error at index: " << i << " should be [" << con[i] << "] but was <" << experimant[i] << ">" << std::endl;
	}
}

int main(int argc, char** argv)
{
	int blockDim = 256;
	int blockNum;
	using namespace std;
	int isEncrypt = 1;
	string pass;
	string dir;
	string dirOut;
	int enlevel;
	cout << "Do you wants to encrypt(1) or decrypt(2)?" << endl;
	cin >> isEncrypt;
	cout << "Enter the directory or file pass you want to en/decrypt: ";
	cin >> dir;
	cout << "Enter the output directory: ";
	cin >> dirOut;
	cout << "Enter the pass word: ";
	cin >> pass;
	cout << "Enter the level of the encryption: ";
	cin >> enlevel;

	if (fs::is_directory(dir)) {
		binFile* files;
		int fileNum = getFileNum(dir);
		__int32*** s = new __int32** [fileNum];
		if (isEncrypt == 1) {
			getFiles(false, files, fileNum, dir);
			for (int i = 0; i < fileNum; i++) {
				blockNum = (files[i].getSize() + blockDim - 1) / blockDim;
				int slength;
				s[i] = rnGen(pass, blockNum, enlevel, slength);
				encrypt(files[i], blockNum, s[i], enlevel, slength);
			}
			writeFiles(true, files, fileNum, dirOut);
		}
		else if (isEncrypt == 2) {
			getFiles(true, files, fileNum, dir);
			for (int i = 0; i < fileNum; i++) {
				blockNum = (files[i].getSize() + blockDim - 1) / blockDim;
				int slength;
				s[i] = rnGen(pass, blockNum, enlevel, slength);
				decrypt(files[i], blockNum, s[i], enlevel, slength);
			}
			writeFiles(false, files, fileNum, dirOut);
		}
	}
	else {
		binFile file;
		int slength;
		__int32** s;
		if (isEncrypt == 1) {
			getFile(false, file, dir);
			blockNum = (file.getSize() + blockDim - 1) / blockDim;
			s = rnGen(pass, blockNum, enlevel, slength);
			encrypt(file, blockNum, s, enlevel, slength);
			writeFile(true, file, dirOut);
		}
		else if (isEncrypt == 2) {
			getFile(true, file, dir);
			blockNum = (file.getSize() + blockDim - 1) / blockDim;
			s = rnGen(pass, blockNum, enlevel, slength);
			decrypt(file, blockNum, s, enlevel, slength);
			writeFile(false, file, dirOut);
		}
	}

	return(0);
}
